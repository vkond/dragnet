#include "hip/hip_runtime.h"
#include "dragnet_gpu.h"

// Read SIGPROC filterbank header
int read_sigproc_header(FILE *file, header* h) {
  int nchar, nbytes=0, expecting_fchannel = 0;
  char string[80];
  float temp;

  // Read header parameters
  for (;;) {
    // Read string size
    strcpy(string,"ERROR");
    fread(&nchar, sizeof(int), 1, file);

    // Skip wrong strings
    if (!(nchar > 1 && nchar < 80)) continue;

    // Increate byte counter
    nbytes += nchar;

    // Read string
    fread(string, nchar, 1, file);
    string[nchar]='\0';

    // Exit at end of header
    if (strcmp(string, "HEADER_END") == 0) break;

    // Read parameters
    if (strcmp(string, "tsamp") == 0) fread(&(h->tsamp), sizeof(double), 1, file);
    else if (strcmp(string,"tstart") == 0) fread(&(h->tstart), sizeof(double), 1, file);
    else if (strcmp(string,"fch1") == 0) fread(&(h->fch1), sizeof(double), 1, file);
    else if (strcmp(string,"foff") == 0) fread(&(h->foff), sizeof(double), 1, file);
    else if (strcmp(string,"nchans") == 0) fread(&(h->nchan), sizeof(int), 1, file);
    else if (strcmp(string,"nifs") == 0) fread(&(h->nif), sizeof(int), 1, file);
    else if (strcmp(string,"nbits") == 0) fread(&(h->nbit), sizeof(int), 1, file);
    else if (strcmp(string,"nsamples") == 0) fread(&(h->nsamp), sizeof(int), 1, file);
    else if (strcmp(string,"az_start") == 0) fread(&(h->az_start), sizeof(double), 1, file);
    else if (strcmp(string,"za_start") == 0) fread(&(h->za_start), sizeof(double), 1, file);
    else if (strcmp(string,"src_raj") == 0) fread(&(h->src_raj), sizeof(double), 1, file);
    else if (strcmp(string,"src_dej") == 0) fread(&(h->src_dej), sizeof(double), 1, file);
    else if (strcmp(string,"telescope_id") == 0) fread(&(h->telescope_id), sizeof(int), 1, file);
    else if (strcmp(string,"machine_id") == 0) fread(&(h->machine_id), sizeof(int), 1, file);
    else if (strcmp(string,"nbeams") == 0) fread(&(h->nbeam), sizeof(int), 1, file);
    else if (strcmp(string,"ibeam") == 0) fread(&(h->ibeam), sizeof(int), 1, file);
    else if (strcmp(string,"source_name") == 0) strcpy(h->source_name, string);
    else if (strcmp(string, "rawdatafile") == 0) strcpy(h->inpfile, string);
    else if (strcmp(string, "FREQUENCY_START") == 0) expecting_fchannel = 1;
    else if (strcmp(string, "FREQUENCY_END") == 0) expecting_fchannel = 0;
    else if ((strcmp(string, "fchannel") == 0) && expecting_fchannel) {
            if (expecting_fchannel == 3) fread(&temp, sizeof(double), 1, file);
            if (expecting_fchannel == 2) {
              fread(&(h->foff), sizeof(double), 1, file);
              h->foff = h->foff - h->fch1;
              expecting_fchannel = 3;
            }
            if (expecting_fchannel == 1) {
              fread(&(h->fch1), sizeof(double), 1, file);
              expecting_fchannel = 2;
            }
    }
  }

  // Get header and buffer sizes
  h->headersize = (int64_t)ftell(file);
  fseek(file, 0, SEEK_END);
  h->buffersize = ftell(file) - h->headersize;
  h->nsamp = h->buffersize/(h->nchan * h->nif * h->nbit/8);

  // Reset file pointer to start of buffer
  rewind(file);
  fseek(file, h->headersize, SEEK_SET);

  return 0;
}

// Get telescope name by its id
static char *telescope_name(int telescope_id) {

   char *telescope, string[80];
   switch (telescope_id) {
   case 0:
      strcpy(string, "Fake");
      break;
   case 1:
      strcpy(string, "Arecibo");
      break;
   case 2:
      strcpy(string, "Ooty");
      break;
   case 3:
      strcpy(string, "Nancay");
      break;
   case 4:
      strcpy(string, "Parkes");
      break;
   case 5:
      strcpy(string, "Jodrell");
      break;
   case 6:
      strcpy(string, "GBT");
      break;
   case 7:
      strcpy(string, "GMRT");
      break;
   case 8:
      strcpy(string, "Effelsberg");
      break;
   case 9:
      strcpy(string, "ATA");
      break;
   case 10:
      strcpy(string, "UTR-2");
      break;
   case 11:
      strcpy(string, "LOFAR");
      break;
   case 12:
      strcpy(string, "FR606");
      break;
   case 13:
      strcpy(string, "DE601");
      break;
   case 14:
      strcpy(string, "UK608");
      break;
   default:
      strcpy(string, "???????");
      break;
   }
   telescope = (char *) calloc(strlen(string) + 1, 1);
   strcpy(telescope, string);
   return telescope;
}

// Get backend name by its id
static char *backend_name(int machine_id) {

   char *backend, string[80];
   switch (machine_id) {
   case 0:
      strcpy(string, "FAKE");
      break;
   case 1:
      strcpy(string, "PSPM");
      break;
   case 2:
      strcpy(string, "WAPP");
      break;
   case 3:
      strcpy(string, "AOFTM");
      break;
   case 4:
      strcpy(string, "BPP");
      break;
   case 5:
      strcpy(string, "OOTY");
      break;
   case 6:
      strcpy(string, "SCAMP");
      break;
   case 7:
      strcpy(string, "SPIGOT");
      break;
   case 10:
      strcpy(string, "ARTEMIS");
      break;
   case 11:
      strcpy(string, "Cobalt");
      break;
   default:
      strcpy(string, "????");
      break;
   }
   backend = (char *) calloc(strlen(string) + 1, 1);
   strcpy(backend, string);
   return backend;
}
 
// Writes out .inf file
void writeinf(header *h, char *outstem, float dm) {

        char outname[1024];
        char tmp1[100], tmp2[100];
        int itmp;
        int ra_h, ra_m, dec_d, dec_m;
        double ra_s, dec_s;
        FILE *infofile;

        sprintf(outname, "%s_DM%.3f.inf", outstem, dm);

        // first check if file already exists. If it does, then return
        // struct stat info;
        // if (stat(outname, &info) == 0) return;

        if ((infofile=fopen(outname, "w")) == NULL) {
                fprintf(stderr, "Error opening output inf-file!\n");
                exit(1);
        }

        fprintf(infofile, " Data file name without suffix          =  %s_DM%.3f\n", outstem, dm);
        fprintf(infofile, " Telescope used                         =  %s\n", telescope_name(h->telescope_id));
        fprintf(infofile, " Instrument used                        =  %s\n", backend_name(h->machine_id));
        fprintf(infofile, " Object being observed                  =  %s\n", h->source_name);
        ra_h = (int) floor(h->src_raj / 10000.0);
        ra_m = (int) floor((h->src_raj - ra_h * 10000) / 100.0);
        ra_s = h->src_raj - ra_h * 10000 - ra_m * 100;
        dec_d = (int) floor(fabs(h->src_dej) / 10000.0);
        dec_m = (int) floor((fabs(h->src_dej) - dec_d * 10000) / 100.0);
        dec_s = fabs(h->src_dej) - dec_d * 10000 - dec_m * 100;
        if (h->src_dej < 0.0) dec_d = -dec_d;
        fprintf(infofile, " J2000 Right Ascension (hh:mm:ss.ssss)  =  %02d:%02d:%02f\n", ra_h, ra_m, ra_s);
        fprintf(infofile, " J2000 Declination     (dd:mm:ss.ssss)  =  %02d:%02d:%s%f\n", dec_d, dec_m, dec_s < 10 ? "0" : "", dec_s);
        fprintf(infofile, " Data observed by                       =  Unknown\n");
        sprintf(tmp1, "%.15f", h->tstart - (int) floor(h->tstart));
        sscanf(tmp1, "%d.%s", &itmp, tmp2);
	fprintf(infofile, " Epoch of observation (MJD)             =  %d.%s\n", (int) floor(h->tstart), tmp2);
	fprintf(infofile, " Barycentered?           (1=yes, 0=no)  =  0\n");
	fprintf(infofile, " Number of bins in the time series      =  %d\n", h->nsamp);
	fprintf(infofile, " Width of each time series bin (sec)    =  %.15g\n", h->tsamp);
	fprintf(infofile, " Any breaks in the data? (1=yes, 0=no)  =  0\n");
	fprintf(infofile, " Type of observation (EM band)          =  Radio\n");
	fprintf(infofile, " Beam diameter (arcsec)                 =  3600\n");
	fprintf(infofile, " Dispersion measure (cm-3 pc)           =  %.12g\n", dm);
	fprintf(infofile, " Central freq of low channel (Mhz)      =  %.12g\n", h->fch1 - (h->nchan - 1) * fabs(h->foff));
	fprintf(infofile, " Total bandwidth (Mhz)                  =  %.12g\n", fabs(h->foff) * h->nchan);
	fprintf(infofile, " Number of channels                     =  %d\n", h->nchan);
	fprintf(infofile, " Channel bandwidth (Mhz)                =  %.12g\n", fabs(h->foff));
	fprintf(infofile, " Data analyzed by                       =  Unknown\n");
	fprintf(infofile, " Any additional notes:\n    Input filterbank samples have %d bits.\n", h->nbit);

        fclose(infofile);
}
