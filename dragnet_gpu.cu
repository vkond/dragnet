#include "hip/hip_runtime.h"
#include "dragnet_gpu.h"

/* ---------- m a i n -----------*/ 

int main(int argc,char *argv[])
{
  int i, out, next;
  header h;
  cmdline opts;
  void *input = 0;
  dedisp_float *output=0;
  dedisp_float *dmlist;
  dedisp_plan plan;
  dedisp_size dm_count, max_delay, nsamp_computed;
  dedisp_error error;
  dedisp_size nbits = 32;
  char *filename, outname[1024];

  // initializing cmdline structure
  opts.device_id = 0;
  opts.verbose = 1;
  strcpy(opts.prefix, "test");
  strcpy(opts.format, "sigproc");
  opts.dm_start = 0.0;
  opts.dm_end = 50.0;
  opts.dm_start = 0.0;
  opts.pulse_width = 4.0;
  opts.dm_tol = 1.25;

  // parsing cmdline
  if (argc == 1) usage(argv[0]);
  next = parse_cmdline(argc, argv, &opts);
  if (argc < next + 1) { 
   fprintf(stderr, "ERROR: No input file!\n");
   return -1;
  }
  filename = argv[next];


  // checking the data format
  if ((strcmp(opts.format, "sigproc")) == 0) {

    dedisp_byte *tmpinput=0;
    FILE *file;

    // reading header and data from the SIGPROC file
    if ((file = fopen(filename, "rb")) == NULL) {
     fprintf(stderr, "Error reading file %s\n", filename);
     return -1;
    }
    // reading Sigproc header
    if ((read_sigproc_header(file, &h)) != 0) {
     fprintf(stderr, "Error reading header from file %s\n", filename);
     return -1;
    }

    // Print information
    if (opts.verbose) {
      printf("----------------------------- INPUT DATA ---------------------------------\n");
      printf("Frequency of highest channel              : %f MHz\n", h.fch1);
      printf("Bandwidth                                 : %f MHz\n", fabs(h.foff)*h.nchan);
      printf("Number of channels (channel width)        : %d (%f MHz)\n", h.nchan,fabs(h.foff));
      printf("Sample time                               : %f us\n", h.tsamp*1e6);
      printf("Observation duration                      : %f s (%d samples)\n", h.tsamp*h.nsamp,h.nsamp);
      printf("Number of polarizations/bit depth         : %d/%d\n", h.nif, h.nbit);
      printf("Input data array size                     : %lu MB\n", h.buffersize/(1<<20));
      printf("Header size                               : %lu bytes\n", h.headersize);
      printf("\n");
    }

    // Exit on wrong type of input data
    if (h.nif != 1) {
      fprintf(stderr, "Wrong number of polarizations (not 1). Exiting.\n");
      return -1;
    }
    if (h.nbit != 8) {
      fprintf(stderr, "Wrong bit depth (not 8). Exiting.\n");
      return -1;
    }

    // Read buffer
    if (opts.verbose) printf("Reading file\n");
    tmpinput=(dedisp_byte *) malloc(sizeof(dedisp_byte)*h.buffersize);
    fread(tmpinput, sizeof(dedisp_byte), h.buffersize, file);
    input = tmpinput;

    // Close file;
    fclose(file);

  } 
  else if ((strcmp(opts.format, "hdf5")) == 0) {

    dedisp_float *tmpinput=0;

    // reading HDF5 input file
    if ((open_hdf5(filename, &h, tmpinput, opts.verbose)) != 0) {
     fprintf(stderr, "Error reading file %s\n", filename);
     return -1;
    }
    input = tmpinput;
  }
  else {
   fprintf(stderr, "ERROR: Can't recognise the format of input data!\n");
   return -1;
  }




  // Intialize GPU
  if (opts.verbose) printf("\nIntializing GPU (device %d)\n", opts.device_id);
  error = dedisp_set_device(opts.device_id);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not set GPU device: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Create a dedispersion plan
  if (opts.verbose) printf("Creating dedispersion plan\n");
  error = dedisp_create_plan(&plan, h.nchan, h.tsamp, h.fch1, h.foff);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not create dedispersion plan: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Generate a list of dispersion measures for the plan
  if (opts.dm_step == 0) {
    if (opts.verbose) printf("Generating optimal DM trials\n");
    error = dedisp_generate_dm_list(plan, opts.dm_start, opts.dm_end, opts.pulse_width, opts.dm_tol);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  } else {
    // Generate a list of dispersion measures for the plan
    if (opts.verbose) printf("Generating linear DM trials\n");
    dm_count=(int) ceil((opts.dm_end - opts.dm_start)/opts.dm_step) + 1;
    dmlist=(dedisp_float *) calloc(sizeof(dedisp_float), dm_count);
    for (i=0; i<dm_count; i++) {
      dmlist[i]=(dedisp_float) opts.dm_start + opts.dm_step*i;
      printf("dm[%d] = %f\n", i, dmlist[i]);
    }
    error=dedisp_set_dm_list(plan, dmlist, dm_count);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  }

  // Get specifics of the computed dedispersion plan
  dm_count = dedisp_get_dm_count(plan);
  max_delay = dedisp_get_max_delay(plan);
  nsamp_computed = h.nsamp-max_delay;
  dmlist=(dedisp_float *)dedisp_get_dm_list(plan);

  // Print information
  if (opts.verbose) {
    printf("----------------------------- DM COMPUTATIONS  ----------------------------\n");
    printf("Computing %ld DMs from %f to %f pc/cm^3\n", dm_count, dmlist[0], dmlist[dm_count-1]);
    printf("Max DM delay is %ld samples (%f seconds)\n", max_delay, max_delay*h.tsamp);
    printf("Computing %ld out of %d total samples (%.2f%% efficiency)\n", nsamp_computed, h.nsamp, 100.0*(dedisp_float)nsamp_computed/h.nsamp);
    if (opts.dm_step==0.0) printf("Pulse width: %f, DM tolerance: %f\n", opts.pulse_width, opts.dm_tol);
    printf("Output data array size : %ld MB\n", (dm_count * nsamp_computed * (nbits/8))/(1<<20));
    printf("\n");
  }

  // Allocate space for the output data
  output=(dedisp_float *)malloc(nsamp_computed * dm_count * nbits/8);
  if (output==NULL) {
    fprintf(stderr, "ERROR: Failed to allocate output array\n");
    return -1;
  }

  // Perform computation
  if (opts.verbose) printf("Dedispersing on the GPU\n");
  clock_t startclock=clock();
  error = dedisp_execute(plan, h.nsamp, (dedisp_byte *)input, h.nbit, (dedisp_byte *)output, nbits, DEDISP_USE_DEFAULT);
  if (error != DEDISP_NO_ERROR) {
    fprintf(stderr, "ERROR: Failed to execute dedispersion plan: %s\n", dedisp_get_error_string(error));
    return -1;
  }
  if (opts.verbose) printf("Dedispersion took %.2f seconds\n",(double)(clock()-startclock)/CLOCKS_PER_SEC);

  // Write output DM trials
  for (i=0; i<dm_count; i++) {
    // Generate output file name
    sprintf(outname, "%s_DM%.3f.dat", opts.prefix, dmlist[i]);
    if ( (out = open(outname, O_TRUNC | O_CREAT | O_WRONLY | O_LARGEFILE, 0644)) == -1 ) {
      fprintf(stderr, "Error opening %s\n", outname);
      return -1;
    }

    // Write buffer
    write(out, output + i*nsamp_computed, sizeof(dedisp_float)*nsamp_computed);
    
    // Close file
    close(out);

    // Write inf file
    writeinf(&h, opts.prefix, dmlist[i]);
  }

  // Clean up
  if (input != NULL) free(input);
  free(output);
  dedisp_destroy_plan(plan);

  return 0;
}
