#include "hip/hip_runtime.h"
#include "dragnet.h"
#include "lofarhdf5.h"
#include "sigproc.h"

/* ---------- m a i n -----------*/ 

int main(int argc,char *argv[])
{
  int out, next;
  unsigned int i;
  header h;
  cmdline opts;
  void *input = 0;
  Input *raw;  // file descriptor of input raw data
  dedisp_float *output=0;
  dedisp_float *dmlist;
  dedisp_plan plan;
  dedisp_size dm_count, max_delay, nsamp_computed;
  dedisp_error error;
  dedisp_size nbits = 32;
  char *filename, outname[1024];

  // initializing cmdline structure
  opts.device_id = 0;
  opts.verbose = 1;
  opts.blocksize = 0;
  strcpy(opts.prefix, "test");
  strcpy(opts.format, "sigproc");
  opts.dm_start = 0.0;
  opts.dm_end = 50.0;
  opts.dm_start = 0.0;
  opts.pulse_width = 4.0;
  opts.dm_tol = 1.25;

  // parsing cmdline
  if (argc == 1) usage(argv[0]);
  next = parse_cmdline(argc, argv, &opts);
  if (argc < next + 1) { 
   fprintf(stderr, "ERROR: No input file!\n");
   return -1;
  }
  filename = argv[next];

  // checking the data format
  if ((strcmp(opts.format, "sigproc")) == 0) {
   // opening the input Sigproc file
   Sigproc infile = Sigproc(filename, &h, opts.verbose);
   raw = &infile;
  } else if ((strcmp(opts.format, "hdf5")) == 0) {
   // opening the input HDF5 file
   HDF5 infile = HDF5(filename, &h, opts.verbose);
   raw = &infile;
  } else {
   fprintf(stderr, "ERROR: Can't recognise the format of input data!\n");
   return -1;
  }

  // checking the block size
  if (opts.blocksize <= 0 || opts.blocksize > h.nsamp) opts.blocksize = h.nsamp;

  // Intialize GPU
  if (opts.verbose) printf("\nIntializing GPU (device %d)\n", opts.device_id);
  error = dedisp_set_device(opts.device_id);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not set GPU device: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Create a dedispersion plan
  if (opts.verbose) printf("Creating dedispersion plan\n");
  error = dedisp_create_plan(&plan, h.nchan, h.tsamp, h.fch1, h.foff);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not create dedispersion plan: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Generate a list of dispersion measures for the plan
  if (opts.dm_step == 0) {
    if (opts.verbose) printf("Generating optimal DM trials\n");
    error = dedisp_generate_dm_list(plan, opts.dm_start, opts.dm_end, opts.pulse_width, opts.dm_tol);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  } else {
    // Generate a list of dispersion measures for the plan
    if (opts.verbose) printf("Generating linear DM trials\n");
    dm_count=(int) ceil((opts.dm_end - opts.dm_start)/opts.dm_step) + 1;
    dmlist=(dedisp_float *) calloc(sizeof(dedisp_float), dm_count);
    for (i=0; i<dm_count; i++) {
      dmlist[i]=(dedisp_float) opts.dm_start + opts.dm_step*i;
      printf("dm[%d] = %f\n", i, dmlist[i]);
    }
    error=dedisp_set_dm_list(plan, dmlist, dm_count);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  }

  // Get specifics of the computed dedispersion plan
  dm_count = dedisp_get_dm_count(plan);
  max_delay = dedisp_get_max_delay(plan);
  nsamp_computed = h.nsamp-max_delay;
  dmlist=(dedisp_float *)dedisp_get_dm_list(plan);

  // Print information
  if (opts.verbose) {
    printf("----------------------------- DM COMPUTATIONS  ----------------------------\n");
    printf("Computing %ld DMs from %f to %f pc/cm^3\n", dm_count, dmlist[0], dmlist[dm_count-1]);
    printf("Max DM delay is %ld samples (%f seconds)\n", max_delay, max_delay*h.tsamp);
    printf("Computing %ld out of %d total samples (%.2f%% efficiency)\n", nsamp_computed, h.nsamp, 100.0*(dedisp_float)nsamp_computed/h.nsamp);
    if (opts.dm_step==0.0) printf("Pulse width: %f, DM tolerance: %f\n", opts.pulse_width, opts.dm_tol);
    printf("Output data array size : %ld MB\n", (dm_count * nsamp_computed * (nbits/8))/(1<<20));
    printf("\n");
  }

  // Loop over data blocks

  int idata = 0; // loop counter
  dedisp_size isamp_computed = 0;
  int64_t read_samples, to_read;
  clock_t startclock;

  do {
      if (opts.verbose) printf("Data block: %d\n", idata);

      to_read = (isamp_computed + opts.blocksize > h.nsamp ? h.nsamp - isamp_computed : opts.blocksize);
      // Reading the input data
      read_samples = raw->read(to_read, max_delay, &h, input);

      // Allocate space for the output data
      output=(dedisp_float *)realloc(output, (to_read - max_delay) * dm_count * nbits/8);
      if (output==NULL) {
        fprintf(stderr, "ERROR: Failed to allocate output array\n");
        return -1;
      }

      // Perform computation
      if (opts.verbose) printf("Dedispersing on the GPU\n");
      startclock=clock();
      error = dedisp_execute(plan, to_read, (dedisp_byte *)input, h.nbit, (dedisp_byte *)output, nbits, DEDISP_USE_DEFAULT);
      if (error != DEDISP_NO_ERROR) {
        fprintf(stderr, "ERROR: Failed to execute dedispersion plan: %s\n", dedisp_get_error_string(error));
        return -1;
      }
      if (opts.verbose) printf("Dedispersion took %.2f seconds\n",(double)(clock()-startclock)/CLOCKS_PER_SEC);

      // Write output DM trials
      for (i=0; i<dm_count; i++) {
        // Generate output file name
        sprintf(outname, "%s_DM%.3f.dat", opts.prefix, dmlist[i]);
        if ( (out = open(outname, (idata == 0 ? O_TRUNC : O_APPEND) | O_CREAT | O_WRONLY | O_LARGEFILE, 0644)) == -1 ) {
          fprintf(stderr, "Error opening %s\n", outname);
          return -1;
        }

        // Write buffer
        write(out, output + i*(to_read - max_delay), sizeof(dedisp_float)*(to_read - max_delay));
    
        // Close file
        close(out);

        // Write inf file
        if (idata == 0) writeinf(&h, opts.prefix, dmlist[i], max_delay);
      }

      isamp_computed += read_samples;
      idata++;
  } while (isamp_computed < nsamp_computed);

  if (opts.verbose) printf("\nFinish.\n");

  // Clean up
  raw->close();
  if (input != NULL) free(input);
  if (output != NULL) free(output);
  dedisp_destroy_plan(plan);

  return 0;
}
