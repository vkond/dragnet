#include "hip/hip_runtime.h"
#include "dragnet.h"
#include "lofarhdf5.h"
#include "sigproc.h"
#include "mask/mask.h"
#include "mask/vectors.h"

/* ---------- m a i n -----------*/ 

int main(int argc,char *argv[])
{
  int out, next;
  unsigned int i;
  header h;
  cmdline opts;
  void *input = 0;
  void *raw;  // file descriptor of input raw data
  dedisp_bool *killmask = NULL;
  dedisp_float *output=0, *finput = 0;
  dedisp_float *dmlist;
  dedisp_plan plan;
  dedisp_size dm_count, max_delay;
  int64_t nsamp_computed;
  dedisp_error error;
  dedisp_size nbits = 32;
  char *filename, outname[1024];
  // rfi masking
  mask obsmask;  // PRESTO type mask struct
  unsigned int is_mask_apply = 0;  // if we want to apply the mask
  float *padvals = NULL;  // padding values for where RFI is (80% of average)
  int *maskchans = NULL;  // masked channels
  int numzapchan, *zapchan = NULL; // user-defined list of channels to zap

  // initializing cmdline structure
  opts.device_id = 0;
  opts.verbose = 1;
  opts.blocksize = 0;
  strcpy(opts.prefix, "test");
  strcpy(opts.format, "sigproc");
  strcpy(opts.maskfile, "\0");
  strcpy(opts.zapchan, "\0");
  opts.dm_start = 0.0;
  opts.dm_end = 50.0;
  opts.pulse_width = 4.0;
  opts.dm_tol = 1.25;
  opts.clip_sigma = 0.0;

  // parsing cmdline
  if (argc == 1) usage(argv[0]);
  next = parse_cmdline(argc, argv, &opts);
  if (argc < next + 1) { 
   fprintf(stderr, "ERROR: No input file!\n");
   return -1;
  }
  filename = argv[next];

  // open the file
  if (raw_open(filename, opts.format, &h, opts.verbose, raw) != 0) 
   exit(-1);

  /* Get list of user-zapped channels */
  if (strcmp(opts.zapchan, "\0") != 0) {
     zapchan = ranges_to_ivect(opts.zapchan, 0, h.nchan - 1, &numzapchan);
     printf ("Number of user-defined channels to zap: %d (%s)\n", numzapchan, opts.zapchan);
  }

  /* Read an input mask if wanted */
  if (strcmp(opts.maskfile, "\0") != 0) {
    is_mask_apply = 1;
    maskchans = gen_ivect(h.nchan);
    read_mask(opts.maskfile, &obsmask);
    printf("Read mask information from '%s'\n\n", opts.maskfile);
    padvals = (float *)malloc(h.nchan * sizeof(float));
    memset(padvals, 0, h.nchan * sizeof(float));
    determine_padvals(opts.maskfile, &obsmask, padvals);
    // inverse channels in the mask
    inverse_mask(&obsmask, padvals);

    if (h.nbit / 8 == 1) {
      finput=(dedisp_float *)malloc((opts.blocksize > h.nsamp ? h.nsamp : opts.blocksize) * h.nchan * sizeof(dedisp_float));
      if (finput==NULL) {
         fprintf(stderr, "ERROR: Failed to allocate finput array\n");
         return -1;
      }
    }
   } else { obsmask.numchan = obsmask.numint = 0; }

  // checking the block size
  if (opts.blocksize <= 0 || opts.blocksize > h.nsamp) opts.blocksize = h.nsamp;

  // Intialize GPU
  if (opts.verbose) printf("\nIntializing GPU (device %d)\n", opts.device_id);
  error = dedisp_set_device(opts.device_id);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not set GPU device: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Create a dedispersion plan
  if (opts.verbose) printf("Creating dedispersion plan...\n");
  error = dedisp_create_plan(&plan, h.nchan, h.tsamp, h.fch1, h.foff);
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Could not create dedispersion plan: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Generate a list of dispersion measures for the plan
  if (opts.dm_step == 0) {
    if (opts.verbose) printf("Generating optimal DM trials...\n");
    error = dedisp_generate_dm_list(plan, opts.dm_start, opts.dm_end, opts.pulse_width, opts.dm_tol);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  } else {
    if (opts.verbose) printf("Generating linear DM trials...\n");
    dm_count=(int) ceil((opts.dm_end - opts.dm_start)/opts.dm_step) + 1;
    dmlist=(dedisp_float *) calloc(sizeof(dedisp_float), dm_count);
    for (i=0; i<dm_count; i++) {
      dmlist[i]=(dedisp_float) opts.dm_start + opts.dm_step*i;
//      printf("dm[%d] = %f\n", i, dmlist[i]);
    }
    error=dedisp_set_dm_list(plan, dmlist, dm_count);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to generate DM list: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  }

  // zapping given channels using Dedisp's killmask
  // freq order is: first chan is high freq
  // VLAD: for whatever reason I couldn't make it work
  // I tried different freqs order and inversed boolean values (0|1)
  /*
  if (strcmp(opts.zapchan, "\0") != 0) {
    killmask = (dedisp_bool *)malloc(sizeof(dedisp_bool) * h.nchan);
    memset(killmask, 1, sizeof(dedisp_bool) * h.nchan);
    for (int jj = 0; jj < numzapchan; jj++) killmask[h.nchan - 1 - zapchan[jj]] = 0;
    error = dedisp_set_killmask(plan, killmask);
    if (error != DEDISP_NO_ERROR) {
      printf("ERROR: Failed to set killmask: %s\n", dedisp_get_error_string(error));
      return -1;
    }
  }
  */

  // Get specifics of the computed dedispersion plan
  dm_count = dedisp_get_dm_count(plan);
  max_delay = dedisp_get_max_delay(plan);
  nsamp_computed = h.nsamp-max_delay;
  dmlist=(dedisp_float *)dedisp_get_dm_list(plan);

  // checking if our blocksize is smaller than max_delay
  if (opts.blocksize <= max_delay) {
      printf("ERROR: input data blocksize (%lld) is smaller than Max DM delay (%ld)!\n", opts.blocksize, max_delay);
      return -1;
  }

  // Print information
  if (opts.verbose) {
    printf("----------------------------- DM COMPUTATIONS  ----------------------------\n");
    printf("Computing %ld DMs from %f to %f pc/cc\n", dm_count, dmlist[0], dmlist[dm_count-1]);
    printf("Max DM delay is %ld samples (%f seconds)\n", max_delay, max_delay*h.tsamp);
    printf("Computing %ld out of %d total samples (%.2f%% efficiency)\n", nsamp_computed, h.nsamp, 100.0*(dedisp_float)nsamp_computed/h.nsamp);
    if (opts.dm_step==0.0) printf("Pulse width: %f, DM tolerance: %f\n", opts.pulse_width, opts.dm_tol);
    printf("Output data array size : %ld MB\n", (dm_count * nsamp_computed * (nbits/8))/(1<<20));
    printf("\n");
  }

  // Setting maximum gulp_size
  dedisp_size gulp_max, gulp_size = dedisp_get_gulp_size(plan);
  if (gulp_size < opts.blocksize - max_delay) gulp_max = gulp_size;
  else gulp_max = opts.blocksize - max_delay;
  printf("Setting gulp_size from %d to %d\n", (int)gulp_size, (int)gulp_max);
  error = dedisp_set_gulp_size(plan, gulp_max);
  //printf("Current gulp_size = %d\n", (int)dedisp_get_gulp_size(plan));
  if (error != DEDISP_NO_ERROR) {
    printf("ERROR: Failed to set gulp_size: %s\n", dedisp_get_error_string(error));
    return -1;
  }

  // Loop over data blocks

  int idata = 0; // loop counter
  int64_t read_samples, to_read, isamp_computed = 0;
  clock_t startclock;

  do {
      if (opts.verbose) printf("Data block: %d\n", idata);

      to_read = (isamp_computed + opts.blocksize > h.nsamp ? h.nsamp - isamp_computed : opts.blocksize);
      // Reading the input data
      read_samples = raw_read(to_read, max_delay, &h, input, raw);
   
      // to zap given channels
      // VLAD: more effectively it would be to use Dedisp's killmask, 
      // but I couldn't make it work (see above)
      if (strcmp(opts.zapchan, "\0") != 0) {
        dedisp_byte *ptr = (dedisp_byte *)input;
        for (int64_t ii=0; ii<to_read; ii++)
          for (int jj=0; jj<numzapchan; jj++) ptr[ii * h.nchan + h.nchan - 1 - zapchan[jj]] = 0;
      }

      // applying rfi mask
      if (is_mask_apply) {
          // filling in finput array
          if (h.nbit / 8 == 1) {
             dedisp_byte *ptr = (dedisp_byte *)input;
             // I need to re-arrange the order of channels
             // as Presto assumed lowest channel to be the first
             for (int64_t ii = 0; ii<to_read * h.nchan; ii++) finput[ii] = ptr[ii];
          } else finput = (dedisp_float *)input;
          apply_mask(finput, &h, to_read, isamp_computed, opts.clip_sigma, padvals, maskchans, &obsmask);
      }

      // Allocate space for the output data
      output=(dedisp_float *)realloc(output, (to_read - max_delay) * dm_count * nbits/8);
      if (output==NULL) {
        fprintf(stderr, "ERROR: Failed to allocate output array\n");
        return -1;
      }

      // Perform computation
      if (opts.verbose) printf("Dedispersing on the GPU\n");
      startclock=clock();
      error = dedisp_execute(plan, to_read, is_mask_apply == 1 ? (dedisp_byte *)finput : (dedisp_byte *)input, \
              is_mask_apply == 1 ? 32 : h.nbit, (dedisp_byte *)output, nbits, DEDISP_USE_DEFAULT);
      if (error != DEDISP_NO_ERROR) {
        fprintf(stderr, "ERROR: Failed to execute dedispersion plan: %s\n", dedisp_get_error_string(error));
        return -1;
      }
      if (opts.verbose) printf("Dedispersion took %.2f seconds\n",(double)(clock()-startclock)/CLOCKS_PER_SEC);

      // Write output DM trials
      for (i=0; i<dm_count; i++) {
        // Generate output file name
        sprintf(outname, "%s_DM%.3f.dat", opts.prefix, dmlist[i]);
        if ( (out = open(outname, (idata == 0 ? O_TRUNC : O_APPEND) | O_CREAT | O_WRONLY | O_LARGEFILE, 0644)) == -1 ) {
          fprintf(stderr, "Error opening %s\n", outname);
          return -1;
        }

        // Write buffer
        write(out, output + i*(to_read - max_delay), sizeof(dedisp_float)*(to_read - max_delay));
    
        // Close file
        close(out);

        // Write inf file
        if (idata == 0) writeinf(&h, opts.prefix, dmlist[i], max_delay);
      }

      isamp_computed += read_samples;
      idata++;
  } while (isamp_computed < nsamp_computed);

  if (opts.verbose) printf("\nFinish.\n");

  // Clean up
  if (is_mask_apply) {
    free(padvals);
    free_mask(obsmask);
    vect_free(maskchans);
    if (h.nbit / 8 == 1) {
     if (finput != NULL) free(finput);
    }
  }
  if (killmask != NULL) free(killmask);
  if (zapchan != NULL) free(zapchan);
  if (input != NULL) free(input);
  if (output != NULL) free(output);
  dedisp_destroy_plan(plan);
  raw_close(raw);

  return 0;
}
