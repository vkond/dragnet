#include "hip/hip_runtime.h"
#include "sigproc.h"

// constructor
Sigproc::Sigproc(char *filename, header* h, int verbose) : Input() {
  // reading Sigproc input file
  if ((open(filename, h, verbose)) != 0) {
     fprintf(stderr, "Error reading file %s\n", filename);
     exit(-1);
  }
}

// closing the file
void Sigproc::close() {
    fclose(fd);
    if (input != NULL) free(input);
}

// destructor
Sigproc::~Sigproc() {
    close();
}

// opens input Sigproc file and read the header
int Sigproc::open(char *filename, header* h, int verbose) {

  current_sample = 0;
  
  // reading header and data from the SIGPROC file
  if ((fd = fopen(filename, "rb")) == NULL) return -1;
  // reading Sigproc header
  if ((read_sigproc_header(fd, h)) != 0) {
    fprintf(stderr, "Error reading header from file %s\n", filename);
    return -1;
  }
  // Print information
  if (verbose) {
    printf("----------------------------- INPUT DATA ---------------------------------\n");
    printf("Frequency of highest channel              : %f MHz\n", h->fch1);
    printf("Bandwidth                                 : %f MHz\n", fabs(h->foff)*h->nchan);
    printf("Number of channels (channel width)        : %d (%f MHz)\n", h->nchan, fabs(h->foff));
    printf("Sample time                               : %f us\n", h->tsamp*1e6);
    printf("Observation duration                      : %f s (%d samples)\n", h->tsamp*h->nsamp, h->nsamp);
    printf("Number of polarizations/bit depth         : %d/%d\n", h->nif, h->nbit);
    printf("Input data array size                     : %lu MB\n", h->buffersize/(1<<20));
    printf("Header size                               : %lu bytes\n", h->headersize);
    printf("\n");
  }

  // Exit on wrong type of input data
  if (h->nif != 1) {
    fprintf(stderr, "Wrong number of polarizations (not 1). Exiting.\n");
    return -1;
  }
  if (h->nbit != 8) {
    fprintf(stderr, "Wrong bit depth (not 8). Exiting.\n");
    return -1;
  }
  return 0;
}

// Read SIGPROC filterbank header
int Sigproc::read_sigproc_header(FILE *file, header* h) {
  int nchar, nbytes=0, expecting_fchannel = 0;
  char string[80];
  float temp;

  // Read header parameters
  for (;;) {
    // Read string size
    strcpy(string,"ERROR");
    fread(&nchar, sizeof(int), 1, file);

    // Skip wrong strings
    if (!(nchar > 1 && nchar < 80)) continue;

    // Increate byte counter
    nbytes += nchar;

    // Read string
    fread(string, nchar, 1, file);
    string[nchar]='\0';

    // Exit at end of header
    if (strcmp(string, "HEADER_END") == 0) break;

    // Read parameters
    if (strcmp(string, "tsamp") == 0) fread(&(h->tsamp), sizeof(double), 1, file);
    else if (strcmp(string,"tstart") == 0) fread(&(h->tstart), sizeof(double), 1, file);
    else if (strcmp(string,"fch1") == 0) fread(&(h->fch1), sizeof(double), 1, file);
    else if (strcmp(string,"foff") == 0) fread(&(h->foff), sizeof(double), 1, file);
    else if (strcmp(string,"nchans") == 0) fread(&(h->nchan), sizeof(int), 1, file);
    else if (strcmp(string,"nifs") == 0) fread(&(h->nif), sizeof(int), 1, file);
    else if (strcmp(string,"nbits") == 0) fread(&(h->nbit), sizeof(int), 1, file);
    else if (strcmp(string,"nsamples") == 0) fread(&(h->nsamp), sizeof(int), 1, file);
    else if (strcmp(string,"az_start") == 0) fread(&(h->az_start), sizeof(double), 1, file);
    else if (strcmp(string,"za_start") == 0) fread(&(h->za_start), sizeof(double), 1, file);
    else if (strcmp(string,"src_raj") == 0) fread(&(h->src_raj), sizeof(double), 1, file);
    else if (strcmp(string,"src_dej") == 0) fread(&(h->src_dej), sizeof(double), 1, file);
    else if (strcmp(string,"telescope_id") == 0) fread(&(h->telescope_id), sizeof(int), 1, file);
    else if (strcmp(string,"machine_id") == 0) fread(&(h->machine_id), sizeof(int), 1, file);
    else if (strcmp(string,"nbeams") == 0) fread(&(h->nbeam), sizeof(int), 1, file);
    else if (strcmp(string,"ibeam") == 0) fread(&(h->ibeam), sizeof(int), 1, file);
    else if (strcmp(string,"source_name") == 0) strcpy(h->source_name, string);
    else if (strcmp(string, "rawdatafile") == 0) strcpy(h->inpfile, string);
    else if (strcmp(string, "FREQUENCY_START") == 0) expecting_fchannel = 1;
    else if (strcmp(string, "FREQUENCY_END") == 0) expecting_fchannel = 0;
    else if ((strcmp(string, "fchannel") == 0) && expecting_fchannel) {
            if (expecting_fchannel == 3) fread(&temp, sizeof(double), 1, file);
            if (expecting_fchannel == 2) {
              fread(&(h->foff), sizeof(double), 1, file);
              h->foff = h->foff - h->fch1;
              expecting_fchannel = 3;
            }
            if (expecting_fchannel == 1) {
              fread(&(h->fch1), sizeof(double), 1, file);
              expecting_fchannel = 2;
            }
    }
  }

  // Get header and buffer sizes
  h->headersize = (int64_t)ftell(file);
  fseek(file, 0, SEEK_END);
  h->buffersize = ftell(file) - h->headersize;
  h->nsamp = h->buffersize/(h->nchan * h->nif * h->nbit/8);

  // Reset file pointer to start of buffer
  rewind(file);
  fseek(file, h->headersize, SEEK_SET);

  return 0;
}

// reading the nsamples from the file
int64_t Sigproc::read(int64_t nsamples, int64_t shift_back, header* h, void*& out) {

  // how many samples we actually read
  int64_t read_samples = current_sample + nsamples > h->nsamp ? h->nsamp - current_sample : nsamples;
  // allocating memory for input buffer
  input=(dedisp_byte *) realloc(input, sizeof(dedisp_byte) * read_samples * h->nchan);

  fread(input, sizeof(dedisp_byte), read_samples * h->nchan, fd);

  out = input; // set the (void*) pointer to where the data are
  current_sample += (read_samples - shift_back);
  // rewinding back by shift_back samples
  fseek (fd, -shift_back * (h->nbit/8) * h->nif * h->nchan, SEEK_CUR);
  return read_samples - shift_back;
}
