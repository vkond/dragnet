#include "hip/hip_runtime.h"
#include "lofarhdf5.h"
#include <iomanip>  // for setprecision use
#include <iostream>

// constructor
HDF5::HDF5(char *filename, header* h, int verbose) : Input() {
  // reading HDF5 input file
  if ((open(filename, h, verbose)) != 0) {
     fprintf(stderr, "Error reading file %s\n", filename);
     exit(-1);
  }
}

// closing the file
void HDF5::close() {
    if (fd != NULL) delete(fd);
    if (stokes != NULL) delete(stokes);
    if (input != NULL) free(input);
}

// destructor
HDF5::~HDF5() {
    close();
}

// opens input HDF5 file and collects all necessary meta data
int HDF5::open(char *filename, header* h, int verbose) {

  current_sample = 0;

  fd = new BF_File (filename);
  cerr << "Reading Lofar HDF5 file..." << endl << endl;
  strcpy(h->inpfile, filename);

  // getting PI
  Attribute<std::string> PI = fd->projectPI();
  if (verbose) if (PI.exists()) cerr << "PI=" << PI.get() << endl;

  // getting project contact
  Attribute<std::string> projectContact = fd->projectContact();
  if (verbose) if (projectContact.exists()) cerr << "PROJECT_CONTACT=" << projectContact.get() << endl;

  // getting targets
  Attribute< std::vector<std::string> > BFtargets = fd->targets();
  if (verbose) {
   if (BFtargets.exists()) {
      std::vector<std::string> t = BFtargets.get();
      std::vector<std::string>::size_type i=0;
      for(std::vector<std::string>::iterator it = t.begin(); it != t.end(); ++it, i++)
        cerr << "TARGET" << i << "=" << *it << endl;
   } else cerr << "TARGET does not exist" << endl;
  }

  // getting frequency center
  Attribute<double> freq = fd->observationFrequencyCenter();
  if (!freq.exists()) {
   cerr << "observationFrequencyCenter not defined" << endl;
   return 1;
  } else if (verbose) cerr << "observation frequency=" << setprecision(20) << freq.get() << endl;

  // getting number of SAPs
  Attribute<unsigned> nsap = fd->observationNofSubArrayPointings();
  if (!nsap.exists()) {
   cerr << "observationNofSubArrayPointings not defined" << endl;
   return 1;
  } else if (verbose) cerr << "number of SAPs=" << nsap.get() << endl;

  // getting  the instance of SAP
  // checking all SAPs if they exist to pick the right one (if there will be two SAPs in one file, only
  // the first one will be picked up)
  unsigned sap_index;
  for (sap_index=0; sap_index<nsap.get(); sap_index++) {
    if (fd->subArrayPointing(sap_index).exists()) break;
  }
  BF_SubArrayPointing sap = fd->subArrayPointing(sap_index);

  Attribute<unsigned> nbeam = sap.observationNofBeams();
  if (!nbeam.exists()) {
   cerr << "sap.observationNofBeams not defined" << endl;
   return 1;
  } else if (verbose) cerr << "number of beams=" << nbeam.get() << endl;

  // getting the instance of first TA beam in the SAP
  // checking all TABs in the SAP if they exist in the file until the first one that exists is found
  unsigned tab_index;
  for (tab_index=0; tab_index<nbeam.get(); tab_index++) {
    if (sap.beam(tab_index).exists()) break;
  }
  BF_BeamGroup beam = sap.beam(tab_index);

  // getting the center frequency of the beam
  Attribute<double> freq2 = beam.beamFrequencyCenter();
  if (!freq2.exists()) {
   cerr << "beam.beamFrequencyCenter not defined" << endl;
   return 1;
  } else if (verbose) cerr << "beam frequency=" << setprecision(20) << freq2.get() << endl;

  // getting the subband width
  Attribute<double> bw2 = beam.subbandWidth();
  if (!bw2.exists()) {
   cerr << "beam.subbandWidth not defined" << endl;
   return 1;
  } else if (verbose) cerr << "sap subbandwidth=" << setprecision(20) << bw2.get() << endl;

  // getting number of channels per subband
  Attribute<unsigned> nchan = beam.channelsPerSubband();
  if (!nchan.exists()) {
   cerr << "beam.channelsPerSubband not defined" << endl;
   return 1;
  } else if (verbose) cerr << "number of channels/sub=" << nchan.get() << endl;

  // getting the pointer for the Stokes class
  stokes = 0;

  for (unsigned i=0; i<4; i++) {
    BF_StokesDataset tmp = beam.stokes(i);
    if (tmp.exists()) {
      stokes = new BF_StokesDataset (beam.stokes(i));
    }
  }

  // getting the Stokes component
  Attribute<std::string> stokesC = stokes->stokesComponent();
  if (verbose) if (stokesC.exists()) cerr << "stokes component=" << stokesC.get() << endl;

  // getting the number of subbands
  Attribute<unsigned> nsub = stokes->nofSubbands();
  if (verbose) { if (nsub.exists()) cerr << "nsub=" << nsub.get() << endl; else cerr << "stokes nofSubbands not defined" << endl; }

  // getting the number of channels for each subband
  Attribute< std::vector<unsigned> > nofchan = stokes->nofChannels();
  if (verbose) if (nchan.exists()) {
                 std::vector<unsigned> nchan = nofchan.get();
                 cerr << "stokes nofChannels size=" << nchan.size() << endl;
                 // for (unsigned i=0; i<nchan.size(); i++) cerr << "stokes nofChannels[" << i << "]=" << nchan[i] << endl;
               } else cerr << "stokes nofChannels not defined" << endl;

  // getting the rank of the dataset
  size_t ndim= stokes->ndims();
  if (verbose) cerr << "stokes ndim=" << ndim << endl;

  if (verbose) {
   std::vector<std::string> files = stokes->externalFiles();
   for (unsigned i=0; i<files.size(); i++)
     cerr << "files[" << i << "]=" << files[i] << endl;
  }

  // getting telescope
  Attribute<std::string> telescope = fd->telescope();
  if (telescope.exists()) {
   if (verbose) cerr << "telescope=" << telescope.get() << endl;
   // setting the telescope
   h->telescope_id = 11;  // For now assuming it is LOFAR 
  }

  // setting machine
  // For now assuming it is LOFAR's COBALT
  h->machine_id = 11;

  // getting the vector of targets
  Attribute< std::vector<std::string> > targets = beam.targets();
  if (targets.exists()) {
    std::vector<std::string> t = targets.get();
    if (t.size() != 0) {
     strcpy(h->source_name, t.front().c_str());
     if (verbose) cerr << "target = " << t.front() << endl;
    } else { if (verbose) cerr << "targets vector is empty" << endl; }
  } else { if (verbose) cerr << "beam target does not exist" << endl; }

  // getting number of samples
  Attribute<unsigned> nsamp = stokes->nofSamples();
  if (nsamp.exists()) h->nsamp = nsamp.get();

  // are data in Complex Voltage format?
  Attribute<bool> volts = beam.complexVoltage();
  if (volts.exists() && volts.get() == 1) {
    cerr << "Can't process complex-voltage data, ndim = " << ndim << endl;
    return 1;
  }
 
  // check for which coordinate is Spectral
  unsigned spectral_dim = 1;

  // getting instance of Coordinates container
  CoordinatesGroup coord = beam.coordinates();
  if (coord.exists()) {
    Attribute< std::vector<std::string> > types = coord.coordinateTypes();
    if (types.exists()) {
      std::vector<std::string> t = types.get();
      for (unsigned i=0; i<t.size(); i++) {
	if (t[i] == "Spectral") {
	  spectral_dim = i;
	  break;
	}
      }
    }
  }

  std::vector<ssize_t> dims = stokes->dims();
  h->nchan = dims[spectral_dim];
  cerr << "Total number of channels=" << h->nchan << endl;
  
  // getting number of Stokes components in one file
  //Attribute<unsigned> npol = beam.nofStokes();
  // getting number of Stokes components in the observation
  Attribute<unsigned> npol = beam.observationNofStokes();
  unsigned stokes_npol = 1;

  if (npol.exists()) stokes_npol = npol.get();

  if (stokes_npol == 1) {
    h->nif = 1;
  } else {
    cerr << "Can't process more than one IFs" << endl;
    return 1;
  }

  h->nbit = 32;
  h->nbeam = 1; // For now assuming there is only one beam
  h->ibeam = 0;

  // getting split Frequency center of the beam
  Attribute<double> cfreq = beam.beamFrequencyCenter();
  if (!cfreq.exists()) {
   cerr << "beamFrequencyCenter not defined" << endl;
   return 1;
  } else { if (verbose) cerr << "beamFrequencyCenter=" << setprecision(20) << cfreq.get() << endl;
  }

  // getting the start MJD
  Attribute<double> mjd = fd->observationStartMJD();
  if (mjd.exists()) h->tstart = mjd.get();
  if (verbose) cerr << "MJD=" << setprecision(20) << h->tstart << endl;

  // getting the clock rate
  Attribute<double> cRate = fd->clockFrequency();
  if (verbose) {
    if (cRate.exists()) cerr << "clockRate=" << setprecision(20) << cRate.get() << endl;
    else cerr << "clockRate undefined" << endl;
  }

  // getting the sampling rate
  Attribute<double> sRate = beam.samplingRate();
  if (verbose) {
    if (sRate.exists()) cerr << "samplingRate=" << setprecision(20) << sRate.get() << endl;
    else cerr << "samplingRate undefined" << endl;
  }

  // getting the sampling time
  Attribute<double> sTime = beam.samplingTime();
  if (sTime.exists()) {
    if (verbose) cerr << "samplingTime=" << setprecision(20) << sTime.get() << " s"<< endl;
    h->tsamp = sTime.get();
  } else if (verbose) cerr << "samplingTime undefined" << endl;

  // getting the channel width
  Attribute<double> rate = beam.channelWidth();
  if (!rate.exists()) {
   cerr << "beam.channelWidth not defined" << endl;
   return 1;
  } else { if (verbose) cerr << "channel Width=" << setprecision(20) << rate.get() << " Hz" << endl;
           h->foff = -1. * rate.get() * 1.0e-6; // We make it negative as is needed by Sigproc, and in MHz !!
         }

  // getting the subband width
  Attribute<double> subwidth = beam.subbandWidth();
  if (verbose) if (subwidth.exists())
                 cerr << "subband Width=" << setprecision(20) << subwidth.get() << " Hz" << endl;
               else cerr << "subband Width undefined" << endl;

  // setting the bandwidth (in MHz) of the file
  double bw_file = h->nchan * rate.get() * 1.0e-6;
  h->fch1 = cfreq.get() + bw_file/2. - fabs(h->foff * 1.0e-6)/2.; // in MHz !

  // getting the RA and DEC of the beam (in degrees)
  Attribute<double> radeg = beam.pointRA();
  if (verbose) {
    if (radeg.exists()) {
      cerr << "RA=" << setprecision(20) << radeg.get() << " deg" << endl;
      int ra_h = (int)(radeg.get()/15.);
      int ra_m = (int)((radeg.get()/15. - ra_h)*60.);
      double ra_s = (radeg.get()/15. - ra_h - ra_m/60.)*3600.;
      char tmp[30];
      sprintf(tmp, "%02d%02d%s%lf", ra_h, ra_m, ra_s < 10 ? "0" : "", ra_s);
      sscanf(tmp, "%lf", &h->src_raj);
      cerr << "RA=" << tmp << endl;
    } else cerr << "RA undefined" << endl;
  }

  Attribute<double> decdeg = beam.pointDEC();
  if (verbose) {
    if (decdeg.exists()) {
      cerr << "DEC=" << setprecision(20) << decdeg.get() << " deg" << endl;
      int dec_d = (int)(fabs(decdeg.get()));
      int dec_m = (int)((fabs(decdeg.get()) - dec_d)*60.);
      double dec_s = (fabs(decdeg.get()) - dec_d - dec_m/60.)*3600.;
      int sign = (int)(decdeg.get());
      if (sign < 0) dec_d = -dec_d;
      char tmp[30];
      sprintf(tmp, "%02d%02d%s%lf", dec_d, dec_m, dec_s < 10 ? "0" : "", dec_s);
      sscanf(tmp, "%lf", &h->src_dej);
      cerr << "DEC=" << tmp << endl;
    } else cerr << "DEC undefined" << endl;
  }

  return 0;
}


// reading the nsamples from the file
int64_t HDF5::read(int64_t nsamples, int64_t shift_back, header* h, void*& out) {

  // how many samples we actually read
  int64_t read_samples = current_sample + nsamples > h->nsamp ? h->nsamp - current_sample : nsamples;
  // allocating memory for input buffer
  input=(dedisp_float *) realloc(input, sizeof(dedisp_float) * read_samples * h->nchan);
  // Reading the data
  vector<size_t> pos (2);
  pos[0] = current_sample;
  pos[1] = 0;

  // I need this because freq-order is different: lowest freq is first in LOFAR data
  // and for dedisp library it's required the highest freq to be the first
  // so we need to re-order
  float *outbuf = new float[read_samples * h->nchan];
  stokes->get2D (pos, outbuf, read_samples, h->nchan);
  for (long ii = 0; ii < read_samples; ii++) {
   for (long jj = 0; jj < h->nchan; jj++) {
    input[(ii+1)*h->nchan-1-jj] = outbuf[ii*h->nchan+jj];
//    cerr << "samp=" << ii << "  chan=" << jj << "  val=" << outbuf[(ii+1)*h->nchan-1-jj] << endl;
   }
  }
  delete(outbuf);

  out = input; // set the (void*) pointer to where the data are
  current_sample += (read_samples - shift_back);
  return read_samples - shift_back;
}
